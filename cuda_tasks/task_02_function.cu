#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define N (1 << 20)

__global__ void kernelB(float *x, float *y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) y[i] = sinf(x[i]) * cosf(x[i]);
}

int main() {
    float *dX, *dY;
    size_t sz = N * sizeof(float);

    hipMalloc(&dX, sz);
    hipMalloc(&dY, sz);

    dim3 block(256), grid((N + block.x - 1) / block.x);
    kernelB<<<grid, block>>>(dX, dY);

    hipFree(dX); hipFree(dY);
    return 0;
}