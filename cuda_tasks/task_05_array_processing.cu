#include <hip/hip_runtime.h>
#include <math.h>

#define N (512 * 50000)

__global__ void processArray(float *A, float *B, float *C) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        float val = A[i] * B[i];
        float sum = 0;
        for (int j = 0; j < 100; ++j) sum += sinf(j + val);
        C[i] = sum;
    }
}

int main() {
    float *dA, *dB, *dC;
    size_t sz = N * sizeof(float);

    hipMalloc(&dA, sz);
    hipMalloc(&dB, sz);
    hipMalloc(&dC, sz);

    dim3 block(256), grid((N + block.x - 1) / block.x);
    processArray<<<grid, block>>>(dA, dB, dC);

    hipFree(dA); hipFree(dB); hipFree(dC);
    return 0;
}