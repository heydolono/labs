#include <hip/hip_runtime.h>
#include <cmath>

#define N 10240
#define EPS 1e-6

__global__ void solveStep(double *A, double *F, double *X0, double *X1) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        double sum = 0;
        for (int j = 0; j < N; ++j) sum += A[j + i * N] * X0[j];
        X1[i] = X0[i] + (F[i] - sum) / A[i + i * N];
    }
}

__global__ void diffVectors(double *X0, double *X1, double *delta) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        delta[i] = fabs(X0[i] - X1[i]);
        X0[i] = X1[i];
    }
}

int main() {
    double *hA = new double[N * N], *hF = new double[N], *hX0 = new double[N], *hX1 = new double[N], *hDelta = new double[N];
    double *dA, *dF, *dX0, *dX1, *dDelta;

    size_t szA = N * N * sizeof(double);
    size_t szX = N * sizeof(double);

    hipMalloc(&dA, szA);
    hipMalloc(&dF, szX);
    hipMalloc(&dX0, szX);
    hipMalloc(&dX1, szX);
    hipMalloc(&dDelta, szX);

    hipMemcpy(dA, hA, szA, hipMemcpyHostToDevice);
    hipMemcpy(dF, hF, szX, hipMemcpyHostToDevice);
    hipMemcpy(dX0, hX0, szX, hipMemcpyHostToDevice);

    int iter = 0;
    double err = 1.0;

    while (err > EPS) {
        solveStep<<<1, N>>>(dA, dF, dX0, dX1);
        diffVectors<<<1, N>>>(dX0, dX1, dDelta);
        hipMemcpy(hDelta, dDelta, szX, hipMemcpyDeviceToHost);
        err = 0;
        for (int i = 0; i < N; ++i) err += hDelta[i];
        err /= N;
        iter++;
    }

    hipFree(dA); hipFree(dF); hipFree(dX0); hipFree(dX1); hipFree(dDelta);
    delete[] hA; delete[] hF; delete[] hX0; delete[] hX1; delete[] hDelta;
    return 0;
}