#include <hip/hip_runtime.h>

#define N 2048
#define BS 32

__global__ void matMul(float *A, float *B, float *C) {
    int row = blockIdx.y * BS + threadIdx.y;
    int col = blockIdx.x * BS + threadIdx.x;
    float sum = 0;
    if (row < N && col < N) {
        for (int k = 0; k < N; ++k)
            sum += A[row * N + k] * B[k * N + col];
        C[row * N + col] = sum;
    }
}

int main() {
    float *dA, *dB, *dC;
    size_t sz = N * N * sizeof(float);

    hipMalloc(&dA, sz);
    hipMalloc(&dB, sz);
    hipMalloc(&dC, sz);

    dim3 block(BS, BS), grid(N / BS, N / BS);
    matMul<<<grid, block>>>(dA, dB, dC);

    hipFree(dA); hipFree(dB); hipFree(dC);
    return 0;
}