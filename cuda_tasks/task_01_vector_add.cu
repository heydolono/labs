#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE (1 << 20)

__global__ void kernelA(float *a, float *b, float *c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < SIZE) c[i] = a[i] + b[i];
}

template<typename T>
void safeFree(T*& ptr) {
    if (ptr) { hipFree(ptr); ptr = nullptr; }
}

int main() {
    float *hA = new float[SIZE], *hB = new float[SIZE], *hC = new float[SIZE];
    float *dA, *dB, *dC;

    hipMalloc(&dA, SIZE * sizeof(float));
    hipMalloc(&dB, SIZE * sizeof(float));
    hipMalloc(&dC, SIZE * sizeof(float));

    for (int i = 0; i < SIZE; ++i) {
        hA[i] = i;
        hB[i] = i * 2;
    }

    hipMemcpy(dA, hA, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, SIZE * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(256), grid((SIZE + block.x - 1) / block.x);
    kernelA<<<grid, block>>>(dA, dB, dC);

    hipMemcpy(hC, dC, SIZE * sizeof(float), hipMemcpyDeviceToHost);

    printf("Result: %f\n", hC[0]);

    delete[] hA; delete[] hB; delete[] hC;
    safeFree(dA); safeFree(dB); safeFree(dC);
    return 0;
}